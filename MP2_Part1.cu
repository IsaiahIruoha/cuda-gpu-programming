#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define TOLERANCE 0.001f          // tolerance for result comparison
#define TILE_WIDTH 16             // change this manually to test different tile sizes

// cpu reference for square matrix multiplication
void matMulCPU(const float* A, const float* B, float* C, int Width)
{
    for (int row = 0; row < Width; row++) {
        for (int col = 0; col < Width; col++) {
            float sum = 0.0f;
            for (int k = 0; k < Width; k++) {
                sum += A[row * Width + k] * B[k * Width + col];
            }
            C[row * Width + col] = sum;
        }
    }
}

// initialize matrix with random floats in [0, 1]
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; i++) {
        data[i] = (float)rand() / (float)RAND_MAX;
    }
}

// compare two arrays elementwise with tolerance
bool compareArrays(const float* ref, const float* gpu, int size, float tolerance = TOLERANCE)
{
    for (int i = 0; i < size; i++) {
        if (fabs(ref[i] - gpu[i]) > tolerance)
            return false;
    }
    return true;
}

// tiled matrix multiplication kernel using static shared memory
__global__ void tiledMatMulKernel_static(const float* M, const float* N, float* P, int Width)
{
    __shared__ float tileM[TILE_WIDTH][TILE_WIDTH];  // static shared memory for M
    __shared__ float tileN[TILE_WIDTH][TILE_WIDTH];  // static shared memory for N

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;  // row index in P
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;  // col index in P

    float Pvalue = 0.0f;
    int numTiles = (Width + TILE_WIDTH - 1) / TILE_WIDTH;

    for (int ph = 0; ph < numTiles; ph++) {
        int tiledCol = ph * TILE_WIDTH + threadIdx.x;  // index for current tile in M
        if (row < Width && tiledCol < Width)
            tileM[threadIdx.y][threadIdx.x] = M[row * Width + tiledCol];
        else
            tileM[threadIdx.y][threadIdx.x] = 0.0f;

        int tiledRow = ph * TILE_WIDTH + threadIdx.y;  // index for current tile in N
        if (tiledRow < Width && col < Width)
            tileN[threadIdx.y][threadIdx.x] = N[tiledRow * Width + col];
        else
            tileN[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; k++) {
            Pvalue += tileM[threadIdx.y][k] * tileN[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < Width && col < Width)
        P[row * Width + col] = Pvalue;
}

int main()
{
    int testSizes[] = {256, 512, 1024, 2048, 4096};
    int numTests = sizeof(testSizes) / sizeof(testSizes[0]);
    srand((unsigned int)time(NULL));

    for (int t = 0; t < numTests; t++) {
        int Width = testSizes[t];
        int size = Width * Width;
        size_t bytes = size * sizeof(float);
        printf("Matrix Multiplication Test: %d x %d\n", Width, Width);

        float* h_M    = (float*)malloc(bytes);
        float* h_N    = (float*)malloc(bytes);
        float* h_P    = (float*)malloc(bytes);
        float* h_Pcpu = (float*)malloc(bytes);

        randomInit(h_M, size);
        randomInit(h_N, size);

        float *d_M, *d_N, *d_P;
        hipMalloc((void**)&d_M, bytes);
        hipMalloc((void**)&d_N, bytes);
        hipMalloc((void**)&d_P, bytes);

        // copy inputs to device (transfer time not measured)
        hipMemcpy(d_M, h_M, bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_N, h_N, bytes, hipMemcpyHostToDevice);

        // set up grid and block dimensions
        dim3 block(TILE_WIDTH, TILE_WIDTH);
        dim3 grid((Width + TILE_WIDTH - 1) / TILE_WIDTH, (Width + TILE_WIDTH - 1) / TILE_WIDTH);

        hipEvent_t start, stop;
        float kernelTime = 0.0f;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // measure kernel execution time
        hipEventRecord(start, 0);
        tiledMatMulKernel_static<<<grid, block>>>(d_M, d_N, d_P, Width);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&kernelTime, start, stop);
        printf("Kernel (GPU) execution time: %.3f ms\n", kernelTime);

        hipMemcpy(h_P, d_P, bytes, hipMemcpyDeviceToHost);

        // cpu reference computation and timing
        clock_t cpuStart = clock();
        matMulCPU(h_M, h_N, h_Pcpu, Width);
        clock_t cpuEnd = clock();
        float cpuTimeMs = 1000.0f * (float)(cpuEnd - cpuStart) / CLOCKS_PER_SEC;
        printf("CPU matrix multiplication time: %.3f ms\n", cpuTimeMs);

        // compare results
        bool correct = compareArrays(h_Pcpu, h_P, size, 1e-3f);
        if (correct)
            printf("Test PASSED for %dx%d!\n\n", Width, Width);
        else
            printf("Test FAILED for %dx%d!\n\n", Width, Width);

        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_M);
        hipFree(d_N);
        hipFree(d_P);
        free(h_M);
        free(h_N);
        free(h_P);
        free(h_Pcpu);
    }
    return 0;
}
